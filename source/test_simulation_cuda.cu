#include "arg_parse.hpp"
#include "simulation_cuda.hpp"
#include "analysis.hpp"
#include "csvr_param.hpp"
#include <limits>
#include <iostream>

#define CPUGPU_ALLOC(type, var, ...) \
  type* var##_ptr; \
  check(hipMallocManaged(&var##_ptr, sizeof(type))); \
  type& var = *(new(var##_ptr) type(__VA_ARGS__))

#define CPUGPU_DELETE(type, var) \
  var.~type(); \
  check(hipFree(&var));

#define check(RESULT) do {                      \
    check(RESULT, __FILE__, __LINE__);          \
  } while(0)

namespace { const char *strerrno(int) { return strerror(errno); } }

template<class T, T Success, const char *(ErrorStr)(T t)>
struct ErrorInfoBase {
  static constexpr bool isSuccess(T t) { return t == Success; }
  static const char *getErrorStr(T t) { return ErrorStr(t); }
};
template<class T> struct ErrorInfo;
template <> struct ErrorInfo<hipError_t> :
  ErrorInfoBase<hipError_t, hipSuccess, hipGetErrorString> {};
template <> struct ErrorInfo<int> :
  ErrorInfoBase<int, 0, strerrno> {};

namespace {
template<class T>
static void (check)(T result, const char *file, unsigned line) {
  if (ErrorInfo<T>::isSuccess(result)) return;
  std::cerr << file << ":"
            << line << ": "
            << ErrorInfo<T>::getErrorStr(result) << "\n";
  exit(-1);
}
}

int main(int argc, char *argv[]) {
    hipSetDevice(1);

    arg_parse::init(argc, argv);
    
    //setting up model
    CPUGPU_ALLOC(model, m, arg_parse::get<bool>("G", "gradients", false),
        arg_parse::get<bool>("P", "perturb", false));
    
    //setting up param_set
    param_set ps;

    csvr_param csvrp(arg_parse::get<string>("p", "param-list", "../models/her_model_2014/param_list.csv"));
    
    if (csvrp.is_open())
    {
        unsigned int set_n = 0;
        while (csvrp.get_next(ps))
        {
            cout << "loaded param_set " << set_n++ << endl;
            CPUGPU_ALLOC(param_set, cudaps);
            cudaps = ps;
            
            //setting up simulation
            RATETYPE analysis_interval = arg_parse::get<RATETYPE>("a","analysis_interval",0.1);

            CPUGPU_ALLOC(simulation_cuda, s, m, cudaps,
                arg_parse::get<int>("c", "cell-total", 200),
                arg_parse::get<int>("w", "total-width", 50),
                arg_parse::get<RATETYPE>("s", "step-size", 0.01),
                analysis_interval,
                arg_parse::get<RATETYPE>("t", "sim_time", 60) );
       
            // DataLogger dl(&s); 
            s.initialize();

            //BasicAnalysis a(&s);
            OscillationAnalysis o(&s,analysis_interval,arg_parse::get<RATETYPE>("r","local_range",4),ph1);
            BasicAnalysis a(&s);
            //run simulation
            s.simulate();
            //s.print_delay()	
            o.test();
            //a.test();
            CPUGPU_DELETE(simulation_cuda, s);
        }
    }
    CPUGPU_DELETE(model, m);
    CPUGPU_DELETE(param_set, ps);
}
