#include "hip/hip_runtime.h"
#include <cmath>
#include "simulation_cuda.hpp"
#include "cell_param.hpp"
#include "model_impl.hpp"
#include "context_determ.hpp"
#include "cuda_context_impl.hpp"
#include <limits>
#include <iostream>

typedef std::numeric_limits<double> dbl;
using namespace std;

void simulation_cuda::initialize(){
    simulation_determ::initialize();
    _baby_cl_cuda.initialize();
}

namespace {
    __global__ void cudasim_execute(simulation_cuda& _sim_cu){

        unsigned int k = threadIdx.x;
        _sim_cu.execute_one(k);
    }
} // end namespace


void simulation_cuda::simulate_cuda(){
    RATETYPE analysis_chunks = time_total/analysis_gran;
    RATETYPE total_step = analysis_gran/_step_size;
    //Set dimensions
    dim3 dimBlock(_cells_total,1,1); //each cell had own thread

    //dim3 dimGrid(1,1,1); //simulation done on single block
    dim3 dimGrid(1,1,1);

    //hipDeviceSetLimit(hipLimitStackSize, 65536);
    //Run kernel
    cout.precision(dbl::max_digits10);
    for (int c=0; c<analysis_chunks; c++){
    	for (int i=0;i<total_step;i++){
        	cout<< _j<< " "<<_baby_cl_cuda[ph11][0][0]<<endl;
        	cudasim_execute<<<dimGrid, dimBlock>>>(*this);

        	hipDeviceSynchronize(); //Required to be able to access managed 
                	                 // GPU data
    	}
    }

    check(hipDeviceSynchronize());
    //convert back to CPU
    if (hipPeekAtLastError() != hipSuccess) {
        cout << "Kernel launch error: " << hipPeekAtLastError() << "\n";
    }

    hipDeviceSynchronize();
}

void simulation_cuda::calc_max_delays() {
  RATETYPE temp_delays[NUM_SPECIES];
  for (int s = 0; s < NUM_SPECIES; s++) {
    max_delays[s] = 0;
    temp_delays[s] = 0.0;
  }
  //for each reaction
  //  for each input
  //    accumulate delay into specie
  //  for each factor
  //    accumulate delay into specie
  //RATETYPE max_gradient_##name = 0; \
  //for (int k = 0; k < _width_total; k++) { \
  //  max_gradient_##name = std::max<int>(_model.factors_gradient[ name ][k], max_gradient_##name); \
  //} 
#define REACTION(name) 
#define DELAY_REACTION(name) \
  for (int in = 0; in < _model.reaction_##name.getNumInputs(); in++) { \
    specie_id sp = _model.reaction_##name.getInputs()[in]; \
    temp_delays[sp] = std::max<RATETYPE>(_parameter_set._delay_sets[ dreact_##name ], temp_delays[sp]); \
  } \
  for (int in = 0; in < _model.reaction_##name.getNumFactors(); in++) { \
    specie_id sp = _model.reaction_##name.getFactors()[in]; \
    temp_delays[sp] = std::max<RATETYPE>(_parameter_set._delay_sets[ dreact_##name ], temp_delays[sp]); \
  }
#include "reactions_list.hpp"
#undef REACTION
#undef DELAY_REACTION
    for (int s = 0; s < NUM_SPECIES; s++) {
        max_delays[s] = temp_delays[s]/_step_size;
    }
}
